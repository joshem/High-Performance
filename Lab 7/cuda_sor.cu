//i am a comment
//this is part 2

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <time.h>
//edit
// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define NUM_THREADS_PER_BLOCK 	256
#define NUM_BLOCKS 				16
#define PRINT_TIME 				1
#define ARR_LEN				2000
#define SM_ARR_LEN          4000000
#define TOL						1e-6
#define ITERS 2000
#define IMUL(a, b) __mul24(a, b)

void initializeArray2D(float *arr, int len, int seed);


__global__ void kernal_SOR(int arrLen, float* x, float* result)
{
	const int row =  threadIdx.x;
	const int column =  threadIdx.y;
	float delta;
	int i,j,count;
	for(count=0;count<=ITERS;count++){//the iterations for loop
		// elemDim/numofThreads  2000/16 
		for(i=row*125;i<(row*125)+125;i++){
			for(j=column*125;j<(column*125)+125;j++){
				if(i>0 && j>0 && i<(ARR_LEN-1) && j<(ARR_LEN-1)){
					delta = x[i * ARR_LEN + j] - 0.25 * (x[(i-1) * ARR_LEN + j] + x[(i+1) * ARR_LEN + j] + x[(i) * ARR_LEN + j - 1] + x[(i) * ARR_LEN+ j + 1]);
					result[i*ARR_LEN + j] = delta;
				}
				else result[i*ARR_LEN+j] = x[i*ARR_LEN + j];
			}
		}
	}
	__syncthreads();

}

int main(int argc, char** argv) {
	int arrLen = 0;
	
	// GPU Timing variables
	hipEvent_t GPU_start, GPU_stop, CPU_start, CPU_stop;
	float elapsed_gpu,elapsed_cpu;
	
	//Arrays on GP global memory
	float *d_x;
	float *d_y;
	float *d_result;

	//Arrays on the host mem
	float *h_x;
	float *h_y;
	float *h_result;
	float *h_result_gold;

	int i, j, errCount = 0, zeroCount = 0;
	

	
	if(argc>1) {
		arrLen = atoi(argv[1]);
	}
	else	{
		arrLen = SM_ARR_LEN;
	}

	CUDA_SAFE_CALL(hipSetDevice(0));
	
	//Allocate GPU memory
	printf("arrlen = %d", arrLen);
	size_t allocSize = arrLen * sizeof(float);
	CUDA_SAFE_CALL(hipMalloc((void **)&d_x, allocSize));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_y, allocSize));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_result, allocSize));

	//Allocate arrays on host memory
	h_x							= (float *) malloc(allocSize);
	//h_y							= (float *) malloc(allocSize);
	h_result					= (float *) malloc(allocSize);
	h_result_gold				= (float *) malloc(allocSize);

	//Init the host arrays
	printf("\nInitializing the arrays ...");
	//Arrays are initialized with a known seed for reproducability
	initializeArray2D(h_x, arrLen, 2453);
	//initializeArray2D(h_y, arrLen, 1467);
	printf("\t... done\n\n");

#if PRINT_TIME
	//CREATE THE CUDA EVENTS
	hipEventCreate(&GPU_start);
	hipEventCreate(&GPU_stop);
	//record event on the default stream
	hipEventRecord(GPU_start, 0);
#endif

	dim3 dimblock(16,16);
	//Transfer the arrays to the GPU memory
	CUDA_SAFE_CALL(hipMemcpy(d_x,h_x, allocSize, hipMemcpyHostToDevice));
	//CUDA_SAFE_CALL(cudaMemcpy(d_y,h_y, allocSize, cudaMemcpyHostToDevice));

	//Launch the kernel
	kernal_SOR<<<1, dimblock>>>(arrLen,d_x,d_result);
	
	//check for errors during launch
	CUDA_SAFE_CALL(hipPeekAtLastError());

	//Transfer the results back to the host
	CUDA_SAFE_CALL(hipMemcpy(h_result, d_result, allocSize, hipMemcpyDeviceToHost));

#if PRINT_TIME
	//stop and destroy timer
	hipEventRecord(GPU_stop,0);
	hipEventSynchronize(GPU_stop);
	hipEventElapsedTime(&elapsed_gpu, GPU_start, GPU_stop);
	printf("\nGPU time: %f (msec)\n", elapsed_gpu);
	hipEventDestroy(GPU_start);
	hipEventDestroy(GPU_stop);
#endif

	//put sor stuff
	float diff;
	int cnt;
	//time the CPU
	hipEventCreate(&CPU_start);
	hipEventCreate(&CPU_stop);
	hipEventRecord(CPU_start,0);
	
	//Compare results
	for(cnt=0;cnt<=ITERS;cnt++){
		for(i=0;i<ARR_LEN;i++){
			for(j=0;j<ARR_LEN;j++){
				if(i>0 && j>0 &&  i < (ARR_LEN-1) && j < (ARR_LEN-1)){//if not on edge
					diff =  h_x[i * ARR_LEN + j] - 0.25 * (h_x[(i-1) * ARR_LEN + j] + h_x[(i+1) * ARR_LEN + j] + h_x[(i) * ARR_LEN + j - 1] + h_x[(i) * ARR_LEN + j + 1]);
                	h_result_gold[i * ARR_LEN + j] = diff;
				}
				else  h_result_gold[i * ARR_LEN + j] = h_x[i * ARR_LEN + j];
			}
		}
	}
#if PRINT_TIME
	//stop and destroy timer
	hipEventRecord(CPU_stop,0);
	hipEventSynchronize(CPU_stop);
	hipEventElapsedTime(&elapsed_cpu, CPU_start, CPU_stop);
	printf("\nCPU time: %f (msec)\n", elapsed_cpu);
	hipEventDestroy(CPU_start);
	hipEventDestroy(CPU_stop);
#endif
	//compare the results
	for(i=0;i<arrLen;i++)
	{
		if(abs(h_result_gold[i] - h_result[i]) > TOL)
			errCount++;
		if(h_result[i] == 0) 
			zeroCount++;
	}
	
	if(errCount>0){
		printf("\nERROR: Failure: %d results didn't match\n", errCount);
	}
	else if(zeroCount>0) printf("\n ERROR: Failure: %d results are zero\n", zeroCount);
	else printf("\nTest Success: All Results Good\n");
	return 0;
}

void initializeArray2D(float *arr, int len, int seed){
	int i,j;
	float randNum;
	srand(seed);
	for(i=0; i<len;i++){
		randNum = (float) rand();
		arr[i] = randNum;
	}

}
